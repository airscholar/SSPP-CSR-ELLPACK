#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers
#include <cstdio>
#include <cstdlib>
#include <utility>
#include "../../mmio.h"
#include "../../MatrixBase.h"
#include "../../OMP/MatrixELLPACK.h"
#include "../../wtime.h"

using namespace std;

inline double dmin(double a, double b) { return a < b ? a : b; }

const int ntimes = 5;

// Simple 1-D thread block
// Size should be at least 1 warp
#define XBD 128
#define YBD 8
const dim3 BLOCK_DIM(XBD, YBD);

// Simple CPU implementation of matrix multiplication.
void CpuMatrixVector(int rows, int *JA, double *AS, int maxNZ, double *x, double *y) {
    double t;
    int i, j;
    for (i = 0; i < rows; i++) {
        t = 0;
        for (j = 0; j < maxNZ; j++) {
            int index = i * maxNZ + j;
            if (index < rows * maxNZ && JA[index] < rows)
                t += AS[index] * x[JA[index]];
        }
        y[i] = t;
    }
}

__device__ void rowReduce(volatile double *sdata, int tid, int s) {
    switch (s) {
        case 16:
            sdata[tid] += sdata[tid + 16];
        case 8:
            sdata[tid] += sdata[tid + 8];
        case 4:
            sdata[tid] += sdata[tid + 4];
        case 2:
            sdata[tid] += sdata[tid + 2];
        case 1:
            sdata[tid] += sdata[tid + 1];
    }
}

__global__ void gpuMatrixVector(int rows, int *JA, double *AS, int maxNZ, double *x, double *y) {
    __shared__ double ax[YBD][XBD];
    int tr = threadIdx.y;
    int tc = threadIdx.x;
    int row = blockIdx.x * blockDim.y + tr;
    int s;
    ax[tr][tc] = 0.0;
    if (row < rows) {
        //ELLPACK Multiplication
        int idxm = row * maxNZ + tc;
        int idxn = row * maxNZ + maxNZ;
        double t1 = 0.0, t2 = 0.0;
        for (int i = idxm; i < idxn; i += XBD * 2) {
            if (i < rows * maxNZ && JA[i] < rows) t1 += AS[i] * x[JA[i]];
            if (i + XBD < idxn) t2 += AS[i + XBD] * x[JA[i + XBD]];

            idxm += XBD * 2;
        }
        ax[tr][tc] = t1 + t2;
    }
    __syncthreads();
    for (s = XBD / 2; s >= 32; s >>= 1) {
        if (tc < s && tc + s < XBD)
            ax[tr][tc] += ax[tr][tc + s];
        __syncthreads();
    }
    s = min(16, XBD / 2);
    if (tc < s) rowReduce(&(ax[tr][0]), tc, s);

    if ((tc == 0) && (row < rows))
        y[row] = ax[tr][tc];
}

int main(int argc, char **argv) {
    int nrows, ncols, nz;

    int *I, *J;
    double *val;

    if (argc < 2) {
        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }

    // read in the matrix
    MatrixBase::readFile(nrows, ncols, nz, I, J, val, argv[1]);

    // sort the matrix
    MatrixBase::sortData(I, J, val, nz);

    // generate a random vector
    double *temp_x = new double[nrows];
    temp_x = MatrixBase::generateVector(nrows);

    // create an ELLPACK matrix
    MatrixELLPACK ellpack(nrows, ncols, nz, I, J, val, temp_x);

    // get the number of non-zero elements per row
    int maxNZ = ellpack.getMaxNZ();

//----------------------- transpose the matrix ----------------------- //
    //transpose JA
    int *h_JA = new int[nrows * maxNZ];
    for (int i = 0; i < nrows; i++) {
        for (int j = 0; j < maxNZ; j++) {
            h_JA[j * nrows + i] = ellpack.getJA()[i * maxNZ + j];
        }
    }

    // transpose AS
    double *h_AS = new double[nrows * maxNZ];
    for (int i = 0; i < nrows; i++) {
        for (int j = 0; j < maxNZ; j++) {
            h_AS[j * nrows + i] = ellpack.getAS()[i * maxNZ + j];
        }
    }

    //update nrows, ncols, nz to reflect the transpose
    int temp = nrows;
    nrows = maxNZ;
    maxNZ = temp;

// ----------------------- Host memory initialisation ----------------------- //
    //  Allocate memory space on the host.
    double *h_x = new double[nrows];
    double *h_y = new double[nz];
    double *h_y_d = new double[nz];

    h_x = MatrixBase::generateVector(nrows);

// ---------------------- Device memory initialisation ---------------------- //
    //  Allocate memory space on the device.
    int *d_JA;
    double *d_x, *d_y, *d_AS;

    // allocate memory for x and y
    checkCudaErrors(hipMalloc((void **) &d_x, nrows * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_y, nz * sizeof(double)));

    // allocate memory for JA and AS
    checkCudaErrors(hipMalloc((void **) &d_JA, nrows * maxNZ * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_AS, nrows * maxNZ * sizeof(double)));

    // Copy matrices from the host (CPU) to the device (GPU).
    checkCudaErrors(hipMemcpy(d_JA, h_JA, nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, h_AS, nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x, nrows * sizeof(double), hipMemcpyHostToDevice));

    // ------------------------ Calculations on the CPU ------------------------- //
    double flopcnt = 2.e-6 * nz;

    // Create the CUDA SDK timer.
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    CpuMatrixVector(nrows, h_JA, h_AS, maxNZ, h_x, h_y);

    timer->stop();
    double cpuflops = flopcnt / timer->getTime();
    double CPUtime = timer->getTime();

// ------------------------ Calculations on the GPU ------------------------- //
    // Calculate the dimension of the grid of blocks (1D) necessary to cover all rows.
    // If the matrix size is less than the block size, then only one block is needed.
    const dim3 GRID_DIM(nrows, 1);

    //print block size and grid size
//    printf("Block size: %d x %d\n", BLOCK_DIM.x, BLOCK_DIM.y);
//    printf("Grid size: %d x %d\n", GRID_DIM.x, GRID_DIM.y);

        double GPUtime = 0;
    double gpuflops = 0;
    for (int i = 0; i < ntimes; i++) {
        timer->reset();
        timer->start();
        gpuMatrixVector<<<GRID_DIM, BLOCK_DIM >>>(nrows, d_JA, d_AS, maxNZ, d_x, d_y);
        checkCudaErrors(hipDeviceSynchronize());
        timer->stop();

        //get errors from kernel
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error name: %s \t Error description: %s \t Error code: %d \t \n", hipGetErrorName(err),
                   hipGetErrorString(err), err);

        //get average time and flops
        GPUtime += timer->getTime();
        gpuflops += flopcnt / timer->getTime();
    }
    GPUtime /= ntimes;
    gpuflops /= ntimes;

    // Download the resulting vector d_y from the device and store it in h_y_d.
    checkCudaErrors(hipMemcpy(h_y_d, d_y, nrows * sizeof(double), hipMemcpyDeviceToHost));

    // Now let's check if the results are the same.
    double reldiff = 0.0f;
    double diff = 0.0f;

    for (int row = 0; row < nrows; ++row) {
        double maxabs = std::max(std::abs(h_y[row]), std::abs(h_y_d[row]));
        if (maxabs == 0.0) maxabs = 1.0;
        reldiff = std::max(reldiff, std::abs(h_y[row] - h_y_d[row]) / maxabs);
        diff = std::max(diff, std::abs(h_y[row] - h_y_d[row]));
    }

    printf("NAME: %-15s TYPE: %-15s OPTION: %-15s CPU_TIME: %-15f GPU_TIME: %-15f CPU_GFLOPS: %-15f GPU_GFLOPS: %-15f MAX_DIFF: %-15f MAX_REL_DIFF: %-15f SPEEDUP: %-15f \n",
           argv[0], argv[1], "ellpack", CPUtime, GPUtime, cpuflops, gpuflops, diff, reldiff, CPUtime / GPUtime);

// ------------------------------- Cleaning up ------------------------------ //

    delete timer;

    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));

//    delete[] h_y_d;
//    delete[] h_JA;
//    delete[] h_AS;
//    delete[] h_x;
//    delete[] h_y;

    return 0;
}