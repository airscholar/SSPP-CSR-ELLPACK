#include "hip/hip_runtime.h"
// 
// Author: Salvatore Filippone salvatore.filippone@cranfield.ac.uk
//

// Computes matrix-vector product. Matrix A is in row-major order
// i.e. A[i, j] is stored in i * ncols + j element of the vector.
//

#include <iostream>
#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers
#include <cstdio>
#include <cstdlib>
#include <utility>
#include "../../mmio.h"
#include "../../MatrixBase.h"
#include "../../OMP/MatrixCSR.h"
#include "../../OMP/MatrixELLPACK.h"
#include "../../wtime.h"

using namespace std;

inline double dmin(double a, double b) { return a < b ? a : b; }

//const int ntimes = 5;

// Simple 1-D thread block
// Size should be at least 1 warp
#define BD 256
const dim3 BLOCK_DIM(BD);

void
readFile(int &M, int &N, int &nz, int *&I, int *&J, double *&val, int &ret_code, MM_typecode &matcode, char *fileName) {
    // Open the file
    FILE *f = fopen(fileName, "r");
    if (f == NULL) {
        printf("Error: could not open file.\n");
        exit(1);
    }

    // Read the Matrix Market banner
    if (mm_read_banner(f, &matcode) != 0) {
        printf("Error: could not process Matrix Market banner.\n");
        exit(1);
    }

    // Check if the matrix type is supported
    if (mm_is_complex(matcode) || !mm_is_matrix(matcode) || !mm_is_sparse(matcode)) {
        printf("Error: unsupported matrix type [%s].\n", mm_typecode_to_str(matcode));
        exit(1);
    }

//    printf("Matrix type: %s \n", mm_typecode_to_str(matcode));
    // Get the size of the sparse matrix
    if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) != 0) {
        printf("Error: could not read matrix size.\n");
        exit(1);
    }

    // Allocate memory for the matrices
    int *tempI = new int[nz];
    int *tempJ = new int[nz];
    double *tempVal = new double[nz];

    // Read the data
    mm_read_mtx_crd_data(f, M, N, nz, tempI, tempJ, tempVal, matcode);

    // Convert the matrix to a symmetric format (if needed)
    int diagonal = 0;
    if (mm_is_symmetric(matcode)) {
        for (int i = 0; i < nz; i++) {
            if (tempI[i] == tempJ[i]) {
                diagonal++;
            }
        }
        int oldNz = nz;
        nz = nz * 2 - diagonal;
        tempI = (int *) realloc(tempI, nz * sizeof(int));
        tempJ = (int *) realloc(tempJ, nz * sizeof(int));
        tempVal = (double *) realloc(tempVal, nz * sizeof(double));
        int k = oldNz;
        for (int i = 0; i < oldNz; i++) {
            if (tempI[i] != tempJ[i]) {
                tempI[k] = tempJ[i];
                tempJ[k] = tempI[i];
                tempVal[k] = tempVal[i];
                k++;
            }
        }
    }

    // assign value to the pointers
    I = new int[nz];
    J = new int[nz];
    val = new double[nz];

    for (int i = 0; i < nz; i++) {
        I[i] = tempI[i];
        J[i] = tempJ[i];
        val[i] = tempVal[i];
    }

    // Free the temporary memory
    delete[] tempI;
    delete[] tempJ;
    delete[] tempVal;

    // Close the file
    fclose(f);
}

// Simple CPU implementation of matrix addition.
// This will be the basis for your implementation.
void CpuMatrixVector(int rows, int *IRP, int *JA, double *AS, double *x, double *y) {
    for (int i = 0; i < rows; i++) {
        double t = 0;
        for (int j = IRP[i]; j < IRP[i + 1]; j++) {
            t += AS[j] * x[JA[j]];
        }
        y[i] = t;
    }
}

void generateVector(int rows, double *A) {
    for (int row = 0; row < rows; row++) {
        A[row] = 1;
    }
}

// GPU implementation of matrix_vector product using a block of threads for
// each row. 
__device__ void rowReduce(volatile double *sdata, int tid) {
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}

__global__ void gpuMatrixVector(int rows, int *IRP, int *JA, double *AS, double *x, double *y) {
    int tid = threadIdx.x;
    int row = blockIdx.x;
    __shared__ double sdata[BD];
    int s = 0;
    sdata[tid] = 0;
    if (row < rows) {
        double sum = 0;
        int start = IRP[row] + tid;
        int end = IRP[row + 1];
        for (int i = start; i < end; i += BD * 2) {
            sum += AS[i] * x[JA[i]];
            if (i + BD < end) {
                sum += AS[i + BD] * x[JA[i + BD]];
            }
        }
        sdata[tid] = sum;
    }
    __syncthreads();

    for (int i = BD / 2; i >= 32; i >>= 1) {
        if (tid < i) {
            sdata[tid] += sdata[tid + i];
        }
        __syncthreads();
    }

    s = min(16, BD / 2);

    if (tid < s) {
        rowReduce(sdata, tid);
    }

    if (tid == 0) {
        y[row] = sdata[0];
    }
}


int main(int argc, char** argv) {
    int nrows, ncols, nz;
    int ret_code;
    MM_typecode matcode;

    int *I, *J;
    double *val;

    if (argc < 2) {
        fprintf(stderr, "Usage: %s [martix-market-filename]\n", argv[0]);
        exit(1);
    }

    MatrixBase::readFile(nrows, ncols, nz, I, J, val, argv[1]);


    MatrixBase::sortData(I, J, val, nz);

    double *h_x = new double[nrows];
    generateVector(nrows, h_x);

    MatrixCSR csr(nrows, ncols, nz, I, J, val, h_x);
// ----------------------- Host memory initialisation ----------------------- //
    //  Allocate memory space on the host.
    int *h_IRP = new int[nrows + 1];
    int *h_JA = new int[nz];
    double *h_AS = new double[nz];
    double *h_y = new double[nz];
    double *h_y_d = new double[nz];

    //IRP
    h_IRP = csr.getIRP();
    //JA
    h_JA = csr.getJA();
    //AS
    h_AS = csr.getAS();

//    //print IRP
//    printf("IRP: ");
//    for (int i = 0; i < nrows + 1; i++) {
//        printf("%d ", h_IRP[i]);
//    }
//    printf("\n");
//
//    //print JA
//    printf("JA: ");
//    for (int i = 0; i < nz; i++) {
//        printf("%d ", h_JA[i]);
//    }
//    printf("\n");
//
//    //print AS
//    printf("AS: ");
//    for (int i = 0; i < nz; i++) {
//        printf("%f ", h_AS[i]);
//    }
//    printf("\n");
//
//    //print x
//    printf("x: ");
//    for (int i = 0; i < nrows; i++) {
//        printf("%f ", h_x[i]);
//    }
//    printf("\n");


// ---------------------- Device memory initialisation ---------------------- //
    //  Allocate memory space on the device.
    int *d_IRP, *d_JA;
    double *d_x, *d_y, *d_AS;

    // allocate memory for A, x and y
    checkCudaErrors(hipMalloc((void **) &d_x, nrows * sizeof(double)));
    checkCudaErrors(hipMalloc((void **) &d_y, nz * sizeof(double)));

    // allocate memory for IRP, JA and AS
    checkCudaErrors(hipMalloc((void **) &d_IRP, (nrows + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_JA, nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_AS, nz * sizeof(double)));

    // Copy matrices from the host (CPU) to the device (GPU).
    checkCudaErrors(hipMemcpy(d_IRP, h_IRP, (nrows + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_JA, h_JA, nz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_AS, h_AS, nz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x, nrows * sizeof(double), hipMemcpyHostToDevice));

    // ------------------------ Calculations on the CPU ------------------------- //
    double flopcnt = 2.e-6 * nz;

    // Create the CUDA SDK timer.
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    timer->start();
    CpuMatrixVector(nrows, h_IRP, h_JA, h_AS, h_x, h_y);

    timer->stop();
    double cpuflops = flopcnt / timer->getTime();
    double CPUtime = timer->getTime();
//    std::cout << "  CPU time: " << timer->getTime() << " ms." << " GFLOPS " << cpuflops << std::endl;

// ------------------------ Calculations on the GPU ------------------------- //

    // Calculate the dimension of the grid of blocks (1D) necessary to cover
    // all rows.
    //use 1 block per row
    const dim3 GRID_DIM((nrows - 1 + BLOCK_DIM.y) / BLOCK_DIM.y, 1);

    //print block size and grid size
//    printf("Block size: %d x %d\n", BLOCK_DIM.x, BLOCK_DIM.y);
//    printf("Grid size: %d x %d\n", GRID_DIM.x, GRID_DIM.y);

    timer->reset();
    timer->start();
    gpuMatrixVector<<<GRID_DIM, BLOCK_DIM >>>(nrows, d_IRP, d_JA, d_AS, d_x, d_y);
    checkCudaErrors(hipDeviceSynchronize());

    timer->stop();
    double gpuflops = flopcnt / timer->getTime();
    double GPUtime = timer->getTime();

//    std::cout << "  GPU time: " << timer->getTime() << " ms." << " GFLOPS " << gpuflops << std::endl;

    // Download the resulting vector d_y from the device and store it in h_y_d.
    checkCudaErrors(hipMemcpy(h_y_d, d_y, nrows * sizeof(double), hipMemcpyDeviceToHost));

//    printf("CPU result \t GPU result \t Difference\n");
//    for (int i = 0; i < nrows; i++) {
//        if(h_y[i] != h_y_d[i])
//            printf("%f \t %f \t %f\n", h_y[i], h_y_d[i], h_y[i] - h_y_d[i]);
//    }

    // Now let's check if the results are the same.
    double reldiff = 0.0f;
    double diff = 0.0f;

    for (int row = 0; row < nrows; ++row) {
        double maxabs = std::max(std::abs(h_y[row]), std::abs(h_y_d[row]));
        if (maxabs == 0.0) maxabs = 1.0;
        reldiff = std::max(reldiff, std::abs(h_y[row] - h_y_d[row]) / maxabs);
        diff = std::max(diff, std::abs(h_y[row] - h_y_d[row]));
//        if (std::abs(h_y[row] - h_y_d[row]) != 0.0)
//            printf("h_y[%d] = %f, h_y_d[%d] = %f, diff = %f\n", row, h_y[row], row, h_y_d[row],
//                   std::abs(h_y[row] - h_y_d[row]));
    }
//    std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;
    // Rel diff should be as close as possible to unit roundoff; double
    // corresponds to IEEE single precision, so unit roundoff is
    // 1.19e-07
    //

    printf("NAME: %-15s CPU_TIME: %-10f  GPU_TIME: %-10f  CPU_GFLOPS: %-10f  GPU_GFLOPS: %-10f  MAX_DIFF: %-10f  MAX_REL_DIFF: %-10f\n", argv[0], CPUtime, GPUtime, cpuflops, gpuflops, diff, reldiff);

// ------------------------------- Cleaning up ------------------------------ //

    delete timer;

    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_IRP));
    checkCudaErrors(hipFree(d_JA));
    checkCudaErrors(hipFree(d_AS));

    delete[] h_y_d;
    delete[] h_IRP;
    delete[] h_JA;
    delete[] h_AS;
    delete[] h_x;
    delete[] h_y;

    return 0;
}